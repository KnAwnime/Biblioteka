
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialGridSamplerBilinear.cu"
#else

static inline void THNN_(SpatialGridSamplerBilinear_shapeCheck)(
    THCState *state,
    THCTensor *input,
    THCTensor *grid,
    THCTensor *gradOutput) {
  THCUNN_argCheck(state, THCTensor_(nDimension)(state, input) == 4, 2, input,
      "4D input tensor expected but got: %s");
  THCUNN_argCheck(state, THCTensor_(nDimension)(state, grid) == 4, 2, grid,
      "4D grid tensor expected but got: %s");

  int nbatch   = THCTensor_(size)(state, input, 0);
  int channels = THCTensor_(size)(state, input, 1);
  int iheight   = THCTensor_(size)(state, input, 2);
  int iwidth    = THCTensor_(size)(state, input, 3);
  int oheight   = THCTensor_(size)(state, grid, 1);
  int owidth    = THCTensor_(size)(state, grid, 2);

  THCUNN_check_dim_size(state, grid, 4, 0, nbatch);
  THCUNN_check_dim_size(state, grid, 4, 3, 2);

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, 4, 0, nbatch);
    THCUNN_check_dim_size(state, gradOutput, 4, 1, channels);
    THCUNN_check_dim_size(state, gradOutput, 4, 2, oheight);
    THCUNN_check_dim_size(state, gradOutput, 4, 3, owidth);
  }
}

TH_API void THNN_(SpatialGridSamplerBilinear_updateOutput)(
    THCState *state,
    THCTensor *input,
    THCTensor *grid,
    THCTensor *output) {

  THCUNN_assertSameGPU(state, 3, input, grid, output);
  THNN_(SpatialGridSamplerBilinear_shapeCheck)(state, input, grid, NULL);
  int N = THCTensor_(size)(state, input, 0);
  int C = THCTensor_(size)(state, input, 1);
  int IH = THCTensor_(size)(state, input, 2);
  int IW = THCTensor_(size)(state, input, 3);
  int H = THCTensor_(size)(state,grid, 1);
  int W = THCTensor_(size)(state, grid, 2);

  // resize output to the same shape as input
  THCTensor_(resize4d)(state, output, N, C, H, W);

  THCDeviceTensor<real, 4> devInput = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> devGrid = toDeviceTensor<real, 4>(state, grid);
  THCDeviceTensor<real, 4> devOutput = toDeviceTensor<real, 4>(state, output);

  int count = N*H*W*2;
  SpatialGridSamplerBilinear_updateOutput_kernel
    <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count, devInput, devGrid, devOutput);
}

TH_API void THNN_(SpatialGridSamplerBilinear_updateGradInput)(
    THCState *state,
    THCTensor *input, THCTensor *gradInput,
    THCTensor *grid, THCTensor *gradGrid,
    THCTensor *gradOutput) {

  THCUNN_assertSameGPU(state, 5, input, gradInput, grid, gradGrid, gradOutput);
  THNN_(SpatialGridSamplerBilinear_shapeCheck)(state, input, grid, gradOutput);
  int N = THCTensor_(size)(state, input, 0);
  int C = THCTensor_(size)(state, input, 1);
  int IH = THCTensor_(size)(state, input, 2);
  int IW = THCTensor_(size)(state, input, 3);
  int H = THCTensor_(size)(state, grid, 1);
  int W = THCTensor_(size)(state, grid, 2);

  THCTensor_(resize4d)(state, gradInput, N, C, IH, IW);
  THCTensor_(resize4d)(state, gradGrid, N, H, W, 2);
  THCTensor_(zero)(state, gradInput);
  THCTensor_(zero)(state, gradGrid);

  THCDeviceTensor<real, 4> devInput = toDeviceTensor<real, 4>(state, input);
  THCDeviceTensor<real, 4> devGradInput = toDeviceTensor<real, 4>(state, gradInput);
  THCDeviceTensor<real, 4> devGrid = toDeviceTensor<real, 4>(state, grid);
  THCDeviceTensor<real, 4> devGradGrid = toDeviceTensor<real, 4>(state, gradGrid);
  THCDeviceTensor<real, 4> devGradOutput = toDeviceTensor<real, 4>(state, gradOutput);

  int count = N*H*W;
  SpatialGridSamplerBilinear_updateGradInput_kernel
    <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count, devInput, devGradInput, devGrid, devGradGrid, devGradOutput);
}

#endif
