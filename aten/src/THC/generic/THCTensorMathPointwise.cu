
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPointwise.cu"
#else

#include <ATen/MemoryOverlap.h>
#include <ATen/NamedTensorUtils.h>

void THCTensor_(cmax)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorMaxOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorMaxOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cmin)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorMinOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorMinOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cmaxValue)(THCState *state, THCTensor *self, THCTensor *src, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self, TensorMaxValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src, TensorMaxValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cminValue)(THCState *state, THCTensor *self, THCTensor *src, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self, TensorMinValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src, TensorMinValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

#if !defined(THC_REAL_IS_BOOL)

static void propagate_names_if_named_tensor_enabled(THCTensor* result, THCTensor* src) {
  at::namedinference::propagate_names(result, src);
}

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(scalar_t* out, scalar_t* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(scalar_t* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));       \
    at::assert_no_internal_overlap(self_);                              \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1<scalar_t>(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
    propagate_names_if_named_tensor_enabled(self_, src);                \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<scalar_t>::exp,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<scalar_t>::sqrt,  Real)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   tan, THCNumerics<scalar_t>::tan,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  atan, THCNumerics<scalar_t>::atan,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   erf, THCNumerics<scalar_t>::erf,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  erfc, THCNumerics<scalar_t>::erfc,  Real)

#endif
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(crossKernel)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, nself, nx, ny, TensorCrossOp<scalar_t>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}

namespace {
c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl> retainTensorImpl(THCTensor* self) {
  c10::raw::intrusive_ptr::incref(self);
  return c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl>::reclaim(self);
}
}

void THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, scalar_t value, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
#ifdef THC_REAL_IS_HALF
  auto alpha = at::Half(value);
#else
  auto alpha = value;
#endif
  at::add_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)), alpha);
}

void THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, scalar_t value, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
#ifdef THC_REAL_IS_HALF
  auto alpha = at::Half(value);
#else
  auto alpha = value;
#endif
  at::sub_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)), alpha);
}

void THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::mul_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

void THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::div_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

void THCTensor_(cfmod)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorCFmodOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorCFmodOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

#endif
#endif
